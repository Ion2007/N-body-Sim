#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "particle.h"
#include "Vector3.h"

// CUDA Kernel
__global__ void computeGravity(Particle* particles, int numParticles, double G, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        Particle& p1 = particles[i];
        Vector3 force(0, 0, 0);

        for (int j = 0; j < numParticles; ++j) {
            if (i == j) continue;
            Particle& p2 = particles[j];
            Vector3 displacement = p2.pos - p1.pos;
            double distance = displacement.magnitude();
            if (distance < 1e-2) continue; // Avoid singularity
            Vector3 direction = displacement / distance;
            double forceMagnitude = (G * p1.mass * p2.mass) / (distance * distance);
            force = force + direction * forceMagnitude;
        }

        // Update velocity and position
        p1.velocity = p1.velocity + force * (dt / p1.mass);
        p1.pos = p1.pos + p1.velocity * dt;
    }
}

extern "C" void updateParticlesCUDA(Particle * particles, int numParticles, double dt, double G) {
    // Allocate memory on the GPU
    Particle* d_particles;
    hipMalloc(&d_particles, numParticles * sizeof(Particle));

    // Copy data from host (CPU) to device (GPU)
    hipMemcpy(d_particles, particles, numParticles * sizeof(Particle), hipMemcpyHostToDevice);

    // Launch the CUDA kernel with an appropriate number of threads and blocks
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    computeGravity << <numBlocks, blockSize >> > (d_particles, numParticles, G, dt);

    // Copy the results back from the GPU to the CPU
    hipMemcpy(particles, d_particles, numParticles * sizeof(Particle), hipMemcpyDeviceToHost);

    // Free the memory on the GPU
    hipFree(d_particles);
}